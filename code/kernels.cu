#include "hip/hip_runtime.h"
#include "kernels.h"
#include "helpers.h"
#include <iostream>
#include <cmath>


__global__
void blur(unsigned char* input_image, unsigned char* output_image, int width, int height) {

    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 5; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                    const int currentoffset = (offset+ox+oy*width)*3;
                    output_red += input_image[currentoffset]; 
                    output_green += input_image[currentoffset+1];
                    output_blue += input_image[currentoffset+2];
                    hits++;
                }
            }
        }
        output_image[offset*3] = output_red/hits;
        output_image[offset*3+1] = output_green/hits;
        output_image[offset*3+2] = output_blue/hits;
    }
}


__global__
void Contour(unsigned char* input_image, unsigned char* output_image, int width, int height) {


    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 1; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                      const int currentoffset = (offset+ox+oy*width)*3;
                    if(ox == 0 && oy == 0){
                       
                        output_red += input_image[currentoffset]*8; 
                        output_green += input_image[currentoffset+1]*8;
                        output_blue += input_image[currentoffset+2]*8;
                    }
                    else{
                        output_red += input_image[currentoffset] * (-1); 
                        output_green += input_image[currentoffset+1] * (-1);
                        output_blue += input_image[currentoffset+2] * (-1);
                   }
                    hits++;
                }
            }
        }

        output_image[offset*3] = output_red;
        output_image[offset*3+1] = output_green;
        output_image[offset*3+2] = output_blue;
    }
}

__global__
void Netteter(unsigned char* input_image, unsigned char* output_image, int width, int height) {


    const unsigned int offset = blockIdx.x*blockDim.x + threadIdx.x;
    int x = offset % width;
    int y = (offset-x)/width;
    int fsize = 1; // Filter size
    if(offset < width*height) {

        float output_red = 0;
        float output_green = 0;
        float output_blue = 0;
        int hits = 0;
        for(int ox = -fsize; ox < fsize+1; ++ox) {
            for(int oy = -fsize; oy < fsize+1; ++oy) {
                if((x+ox) > -1 && (x+ox) < width && (y+oy) > -1 && (y+oy) < height) {
                      const int currentoffset = (offset+ox+oy*width)*3;
                    if(ox == 0 && oy == 0){
                       
                        output_red += input_image[currentoffset]*5; 
                        output_green += input_image[currentoffset+1]*5;
                        output_blue += input_image[currentoffset+2]*5;
                    }
                    else  if(ox == 0 || oy == 0){
                       
                        output_red -= input_image[currentoffset]; 
                        output_green -= input_image[currentoffset+1];
                        output_blue -= input_image[currentoffset+2];
                    }
                    else{
                        output_red += 0; 
                        output_green += 0;
                        output_blue += 0;
                   }
                    hits++;
                }
            }
        }

        output_image[offset*3] = output_red;
        output_image[offset*3+1] = output_green;
        output_image[offset*3+2] = output_blue;
    }
}





//extern "C" 
void filter (unsigned char* input_image, unsigned char* output_image, int width, int height, int idSelection) {

    unsigned char* dev_input;
    unsigned char* dev_output;
    getError(hipMalloc( (void**) &dev_input, width*height*3*sizeof(unsigned char)));
	
	size_t intval= width*height*3*sizeof(unsigned char);
	
    getError(hipMemcpy( dev_input, input_image, intval, hipMemcpyHostToDevice ));
//	CUDA_SAFE_CALL(hipMemcpy( dev_input, input_image, intval, hipMemcpyHostToDevice )) //CUDA<=5
///	printf("size=%zd\n",intval);
 
    getError(hipMalloc( (void**) &dev_output, width*height*3*sizeof(unsigned char)));

///    dim3 blockDims(512,1,1);
    dim3 blockDims(1024,1,1);
    dim3 gridDims((unsigned int) ceil((double)(width*height*3/blockDims.x)), 1, 1 );


    if(idSelection == 0){
        blur<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 
    }
    if(idSelection == 1){
        Contour<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 
    }
      if(idSelection == 2){
        Netteter<<<gridDims, blockDims>>>(dev_input, dev_output, width, height); 
    }
      printf ("\n finie \n");
    getError(hipMemcpy(output_image, dev_output, width*height*3*sizeof(unsigned char), hipMemcpyDeviceToHost ));

    getError(hipFree(dev_input));
    getError(hipFree(dev_output));

}

